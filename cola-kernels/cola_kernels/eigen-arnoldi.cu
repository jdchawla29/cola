#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <time.h>

typedef std::complex<float> cfloat;
#define index(i, j, N) ((i) * (N)) + (j)

// Kernel declarations
__global__ void sum(hipFloatComplex *v, hipFloatComplex *v_r, int N);
__global__ void sum_of_squares(hipFloatComplex *v, hipFloatComplex *v_r, int N);
__global__ void norm_cal(hipFloatComplex *v, hipFloatComplex norm, int n);
__global__ void update_arr(hipFloatComplex *Q, hipFloatComplex *V, int N, int M, int i);
__global__ void matrix_vector_mul(float *A, hipFloatComplex *B, hipFloatComplex *C, int N, int M, int i);
__global__ void update_new_vector(hipFloatComplex A, hipFloatComplex *B, hipFloatComplex *C, int N, int M, int i);
__global__ void angle_cal(hipFloatComplex *Q, hipFloatComplex *V, hipFloatComplex *ang, int N, int M, int i);

int main(int argc, char *argv[])
{
    time_t start_t, end_t;
    double diff_t;
    int idx = 0;
    cfloat norm, angle;
    srand(21);
    int random_number = rand();

    unsigned int N = atoi(argv[1]);
    int max_iters = atoi(argv[2]);
    int blocks = (N + 1023) / 1024;

    // cpu data
    float *A;
    cfloat *new_vector, *new_vector_r, *H, *Q, *h_vec, *ang;
    A = (float *)calloc(N * N, sizeof(float));
    new_vector = (cfloat *)calloc(N, sizeof(cfloat));
    new_vector_r = (cfloat *)calloc(blocks, sizeof(cfloat));
    H = (cfloat *)calloc((max_iters + 1) * max_iters, sizeof(cfloat));
    Q = (cfloat *)calloc(N * (max_iters + 1), sizeof(cfloat));
    h_vec = (cfloat *)calloc((max_iters + 1), sizeof(cfloat));
    ang = (cfloat *)calloc(N, sizeof(cfloat));

    // cuda data
    float *A_d;
    hipFloatComplex norm_d, angle_d;
    hipFloatComplex *new_vector_d, *new_vector_d_r, *H_d, *Q_d, *h_vec_d, *ang_d;
    hipMalloc((void **)&A_d, N * N * sizeof(float));
    hipMalloc((void **)&new_vector_d, N * sizeof(hipFloatComplex));
    hipMalloc((void **)&new_vector_d_r, blocks * sizeof(hipFloatComplex));
    hipMalloc((void **)&H_d, (max_iters + 1) * max_iters * sizeof(hipFloatComplex));
    hipMalloc((void **)&Q_d, N * (max_iters + 1) * sizeof(hipFloatComplex));
    hipMalloc((void **)&h_vec_d, (max_iters + 1) * sizeof(hipFloatComplex));
    hipMalloc((void **)&ang_d, N * sizeof(hipFloatComplex));

    // initialization
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[index(i, j, N)] = (float)(rand() % 100);
            // A[index(i, j, N)] = float(i*N+j), 0.0f;
        }
    }

    for (int i = 0; i < N; i++)
    {
        // new_vector[i] = cfloat(float(i), 0.0f);
        new_vector[i] = ((float)rand() / RAND_MAX);
    }

    time(&start_t);

    // cuda mem copy
    hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(new_vector_d, new_vector, N * sizeof(cfloat), hipMemcpyHostToDevice);
    hipMemcpy(new_vector_d_r, new_vector_r, N * sizeof(cfloat), hipMemcpyHostToDevice);
    hipMemcpy(H_d, H, max_iters * (max_iters + 1) * sizeof(cfloat), hipMemcpyHostToDevice);
    hipMemcpy(Q_d, Q, N * (max_iters + 1) * sizeof(cfloat), hipMemcpyHostToDevice);
    hipMemcpy(ang_d, ang, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(h_vec_d, h_vec, (max_iters + 1) * sizeof(float), hipMemcpyHostToDevice);

    // norm calcuations
    sum_of_squares<<<blocks, 1024>>>(new_vector_d, new_vector_d_r, N);
    hipMemcpy(new_vector_r, new_vector_d_r, blocks * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    norm = 0;
    for (int i = 0; i < blocks; i++)
    {
        norm = norm + new_vector_r[i];
    }
    norm = sqrt(norm);
    norm_d = make_hipFloatComplex(norm.real(), norm.imag());
    std::cout << "norm(" << norm.real() << ", " << norm.imag() << "i)" << std::endl;
    norm_cal<<<(N + 1023) / 1024, 1024>>>(new_vector_d, norm_d, N);

    // update Q
    update_arr<<<(N + 1023) / 1024, 1024>>>(Q_d, new_vector_d, N, (max_iters + 1), 0);

    // iterations
    while (idx < max_iters)
    {
        matrix_vector_mul<<<blocks, 1024>>>(A_d, Q_d, new_vector_d, N, (max_iters + 1), idx);
        hipMemset(h_vec_d, 0, (max_iters + 1) * sizeof(hipFloatComplex));
        for (int j = 0; j < idx + 1; j++)
        {
            hipMemset(ang_d, 0, N * sizeof(hipFloatComplex));
            hipMemset(new_vector_d_r, 0, blocks * sizeof(hipFloatComplex));
            memset(new_vector_r, 0, blocks * sizeof(cfloat));
            angle_cal<<<blocks, 1024>>>(Q_d, new_vector_d, ang_d, N, (max_iters + 1), j);
            sum<<<blocks, 1024>>>(ang_d, new_vector_d_r, N);
            hipMemcpy(new_vector_r, new_vector_d_r, blocks * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
            angle = 0;
            for (int i = 0; i < blocks; i++)
            {
                angle = angle + new_vector_r[i];
            }
            angle_d = make_hipFloatComplex(angle.real(), angle.imag());
            std::cout << "angle(" << angle.real() << ", " << angle.imag() << "i)" << std::endl;
            hipMemcpy(&h_vec_d[j], &angle_d, sizeof(hipFloatComplex), hipMemcpyHostToDevice);
            update_new_vector<<<blocks, 1024>>>(angle_d, Q_d, new_vector_d, N, (max_iters + 1), j);
        }
        hipMemset(new_vector_d_r, 0, blocks * sizeof(hipFloatComplex));
        memset(new_vector_r, 0, blocks * sizeof(cfloat));
        sum_of_squares<<<blocks, 1024>>>(new_vector_d, new_vector_d_r, N);
        hipMemcpy(new_vector_r, new_vector_d_r, blocks * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        norm = 0;
        for (int i = 0; i < blocks; i++)
        {
            norm = norm + new_vector_r[i];
        }
        norm = sqrt(norm);
        norm_d = make_hipFloatComplex(norm.real(), norm.imag());
        std::cout << "norm(" << norm.real() << ", " << norm.imag() << "i)" << std::endl;
        if (std::abs(norm.real()) < 0.0000001 / 2.0)
        {
            norm.real(0.0000001 / 2.0);
            norm.imag(0.0f);
        }
        norm_cal<<<(N + 1023) / 1024, 1024>>>(new_vector_d, norm_d, N);
        hipMemcpy(&h_vec_d[idx + 1], &norm_d, sizeof(hipFloatComplex), hipMemcpyHostToDevice);
        update_arr<<<((max_iters + 1) + 1023) / 1024, 1024>>>(H_d, h_vec_d, (max_iters + 1), max_iters, idx);
        update_arr<<<(N + 1023) / 1024, 1024>>>(Q_d, new_vector_d, N, (max_iters + 1), (idx + 1));
        idx += 1;
    }

    hipMemcpy(H, H_d, (max_iters + 1) * max_iters * sizeof(cfloat), hipMemcpyDeviceToHost);
    hipMemcpy(Q, Q_d, N * (max_iters + 1) * sizeof(cfloat), hipMemcpyDeviceToHost);

    time(&end_t);
    diff_t = difftime(end_t, start_t);
    printf("Elapsed wall-clock time: %f seconds\n", diff_t);
}

// cuda functions

__global__ void sum(hipFloatComplex *array, hipFloatComplex *out, int size)
{
    __shared__ hipFloatComplex sharedMem[1024]; // Shared memory for block reduction

    int tid = threadIdx.x;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    sharedMem[tid] = (globalIdx < size) ? array[globalIdx] : make_hipFloatComplex(0.0f, 0.0f);
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sharedMem[tid] = hipCaddf(sharedMem[tid], sharedMem[tid + stride]);
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        out[blockIdx.x] = sharedMem[0];
    }
}

__global__ void sum_of_squares(hipFloatComplex *array, hipFloatComplex *out, int size)
{
    __shared__ hipFloatComplex sharedMem[1024]; // Shared memory for block reduction

    int tid = threadIdx.x;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    sharedMem[tid] = (globalIdx < size) ? hipCmulf(array[globalIdx], array[globalIdx]) : make_hipFloatComplex(0.0f, 0.0f);
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sharedMem[tid] = hipCaddf(sharedMem[tid], sharedMem[tid + stride]);
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        out[blockIdx.x] = sharedMem[0];
    }
}

__global__ void norm_cal(hipFloatComplex *v, hipFloatComplex norm, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        v[tid] = hipCdivf(v[tid], norm);
    }
}

__global__ void update_arr(hipFloatComplex *Q, hipFloatComplex *V, int N, int M, int i)
{
    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_idx < N)
    {
        Q[index(row_idx, i, M)] = V[row_idx];
    }
}

__global__ void matrix_vector_mul(float *A, hipFloatComplex *B, hipFloatComplex *C, int N, int M, int i)
{
    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_idx < N)
    {
        hipFloatComplex dot_product = make_hipFloatComplex(0.0f, 0.0f);
        for (int col_idx = 0; col_idx < N; col_idx++)
        {
            hipFloatComplex a = make_hipFloatComplex(A[index(row_idx, col_idx, N)], 0.0f);
            dot_product = hipCaddf(dot_product, hipCmulf(a, B[index(col_idx, i, M)]));
        }
        C[row_idx] = dot_product;
    }
}

__global__ void update_new_vector(hipFloatComplex A, hipFloatComplex *B, hipFloatComplex *C, int N, int M, int i)
{
    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_idx < N)
    {
        C[row_idx] = hipCsubf(C[row_idx], hipCmulf(A, B[index(row_idx, i, M)]));
    }
}

__global__ void angle_cal(hipFloatComplex *Q, hipFloatComplex *V, hipFloatComplex *ang, int N, int M, int i)
{
    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_idx < N)
    {
        ang[row_idx] = hipCmulf(hipConjf(Q[index(row_idx, i, M)]), V[row_idx]);
    }
}

// cudaMemset(ang_d, 0, N * sizeof(cuFloatComplex));
// cudaMemset(new_vector_d_r, 0, blocks * sizeof(cuFloatComplex));
// memset(new_vector_r, 0, blocks * sizeof(cfloat));

// cudaMemcpy(new_vector, new_vector_d, N * sizeof(cfloat), cudaMemcpyDeviceToHost);
// for (int i = 0; i < N; i++)
// {
//     std::cout << "(" << new_vector[i].real() << ", " << new_vector[i].imag() << "i)" << std::endl;
// }

// for (int i = 0; i < max_iters + 1; i++)
// {
//     for (int j = 0; j < max_iters; j++)
//     {
//         std::cout << "(" << H[index(i, j, max_iters)].real() << ", " << H[index(i, j, max_iters)].imag() << "i)";
//     }
//     printf("\n");
// }

// for (int i = 0; i < N; i++)
// {
//     for (int j = 0; j < max_iters + 1; j++)
//     {
//         std::cout << "(" << Q[index(i, j, max_iters+1)].real() << ", " << Q[index(i, j, max_iters+1)].imag() << "i)";
//     }
//     printf("\n");
// }