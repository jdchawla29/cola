#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

namespace cola_kernels {

__global__ void fuse_kernel_1(int numel, const float* a, const float* b, const float* c, float* result) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numel) result[idx] = a[idx] * b[idx] + c[idx];
}

at::Tensor fuse_kernel_1_cuda(const at::Tensor& a, const at::Tensor& b, const at::Tensor& c) {
  TORCH_CHECK(a.sizes() == b.sizes());
  TORCH_CHECK(a.sizes() == c.sizes());
  TORCH_CHECK(a.dtype() == at::kFloat);
  TORCH_CHECK(b.dtype() == at::kFloat);
  TORCH_CHECK(c.dtype() == at::kFloat);
  TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(b.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(c.device().type() == at::DeviceType::CUDA);
  at::Tensor a_contig = a.contiguous();
  at::Tensor b_contig = b.contiguous();
  at::Tensor c_contig = c.contiguous();
  at::Tensor result = torch::empty(a_contig.sizes(), a_contig.options());
  const float* a_ptr = a_contig.data_ptr<float>();
  const float* b_ptr = b_contig.data_ptr<float>();
  const float* c_ptr = c_contig.data_ptr<float>();
  float* result_ptr = result.data_ptr<float>();

  int numel = a_contig.numel();
  fuse_kernel_1<<<(numel+255)/256, 256>>>(numel, a_ptr, b_ptr, c_ptr, result_ptr);
  return result;
}

#define MATIDX(i,j,N) (j * N + i)

__global__ void decompose_cholesky_mm_kernel_device(float *a, int N) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x; // Global thread index
    cg::grid_group grid = cg::this_grid();

    // Loop over k
    for (int k = 0; k < N; k++) {

        // Compute diagonal element in the first thread
        if (thread_id == 0) {
            //printf("sqrt: %f\n", a[MATIDX(k, k, N)]);
            a[MATIDX(k, k, N)] = sqrt(a[MATIDX(k, k, N)]);

            // Update column elements by dividing by the diagonal
            for (int j = k + 1; j < N; j++) {
                //printf("div: %f %f\n", a[MATIDX(j, k, N)], a[MATIDX(k, k, N)]);
                a[MATIDX(j, k, N)] /= a[MATIDX(k, k, N)];
            }
        }

        grid.sync(); // Synchronize threads after updating the diagonal

        // Update the rest of the matrix, only threads that handle i > k
        int i = thread_id + k + 1; // Global row index
        if (i < N) {
            for (int j = i; j < N; j++) {
                //printf("parallel: %f %f %f\n", a[MATIDX(i, j, N)], a[MATIDX(i, k, N)], a[MATIDX(j, k, N)]);
                a[MATIDX(j, i, N)] -= a[MATIDX(i, k, N)] * a[MATIDX(j, k, N)];
            }
        }

        grid.sync(); // Synchronize threads after updating the matrix
    }

    // Zero out the upper triangular part of the matrix after decomposition (only for thread 0)
    if (thread_id == 0) {
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < i; j++) {
                a[MATIDX(j, i, N)] = 0;
            }
        }
    }
    // int row = thread_id / N;
    // int col = thread_id % N;

    // if (row < N && col < N && col > row) {
    //     a[MATIDX(row, col, N)] = 0;
   // }
}
__global__
void inverse_lower_mm_kernel_device(float *a, float *aInv, int N) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x; // Global thread index
    cg::grid_group grid = cg::this_grid();

    if (threadIdx.x == 0) {
        // Initialize `aInv` to zero
        for (int i = 0; i < N; i++) {
            for (int j = 0; j <= i; j++) {
                aInv[MATIDX(i, j, N)] = 0;
            }
        }
    }

    // int i = thread_id / N; // Row index
    // int j = thread_id % N; // Column index

    // // Check if the thread corresponds to a lower triangular element (including diagonal)
    // if (i < N && j <= i) {
    //     aInv[MATIDX(i, j, N)] = 0;
    // }

   grid.sync();

    // Compute the elements of the lower inverse matrix
    for (int j = 0; j < N; j++) {
        for (int i = j + 1; i < N; i++) {

            if (thread_id == 0) {
                aInv[MATIDX(i, j, N)] = -a[MATIDX(i, j, N)] / 
                                                    (a[MATIDX(j, j, N)] * a[MATIDX(i, i, N)]);
            }
            grid.sync();

            int k = thread_id + j + 1;
            if (k < i) {
                atomicAdd((float*)&aInv[MATIDX(i, j, N)], 
                          -a[MATIDX(i, k, N)] * aInv[MATIDX(k, j, N)] / a[MATIDX(i, i, N)]);
            }
        }
    }
    grid.sync();

    if (thread_id == 0) {
        // Set the inverse of the diagonal elements and copy results to `aInv`
        for (int i = 0; i < N; i++) {
            aInv[MATIDX(i, i, N)] = 1.0 / a[MATIDX(i, i, N)];
            for (int j = 0; j <= i; j++) {
                // Set only the lower triangular values in `aInv`
                aInv[MATIDX(i, j, N)] = aInv[MATIDX(i, j, N)];
            }
        }
    }
}

__global__
void multiply_lower_mm_kernel_device(float *a, float *aInv, int N) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x; // Global thread index
    cg::grid_group grid = cg::this_grid();

    // Perform multiplication directly in `aInv`
    for (int j = 0; j < N; j++) {
        for (int i = j; i < N; i++) {

            if (thread_id == 0) {
                aInv[MATIDX(i, j, N)] *= aInv[MATIDX(i, i, N)];
            }
            grid.sync();

            int k = thread_id + i + 1;
            if (k < N) {
                atomicAdd((float*)&aInv[MATIDX(i, j, N)], 
                          aInv[MATIDX(k, j, N)] * aInv[MATIDX(k, i, N)]);
            }
        }
    }
    grid.sync();

    if (thread_id == 0) {
        // Copy the results into the full lower and symmetric upper triangle of `aInv`
        for (int i = 0; i < N; i++) {
            for (int j = 0; j <= i; j++) {
                aInv[MATIDX(j, i, N)] = aInv[MATIDX(i, j, N)];
            }
        }
    }
    // int i = thread_id / N; // Row index
    // int j = thread_id % N; // Column index

    // // Ensure the thread is within bounds and in the upper triangular part (j <= i)
    // if (i < N && j <= i) {
    //     aInv[MATIDX(j, i, N)] = aInv[MATIDX(i, j, N)];
    // }
}


at::Tensor fuse_kernel_2_cuda(at::Tensor& a){
    TORCH_CHECK(a.dtype() == at::kFloat);
    TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
    at::Tensor a_contig = a.contiguous();
    at::Tensor result = torch::empty(a_contig.sizes(), a_contig.options());
    float* a_ptr = a_contig.data_ptr<float>();
    float* result_ptr = result.data_ptr<float>();

    int N = static_cast<int>(a.size(0));
    // printf("N = %d\n", N);
    // TEST
    // at::Tensor a_cpu = a.cpu();  
    // // Ensure it is contiguous on the CPU
    // at::Tensor a_contig_cpu = a_cpu.contiguous();
    // float* a_ptr_cpu = a_contig_cpu.data_ptr<float>();
    // for (int i = 0; i < 3; ++i) {
    //     for (int j = 0; j < 3; ++j) {
    //         float value = a_ptr_cpu[i * 3 + j];
    //         printf("A[%d, %d] = %f\n", i, j, value);
    //     }
    // }
    //OVER

    // decompose_cholesky_mm_kernel_device<<< 1, N>>>(a_ptr, N);

    // hipDeviceSynchronize();
    //Kernel launch configuration for cooperative kernel
    dim3 blockSize(256);  // Number of threads per block
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);  // Number of blocks
    // Kernel launch with cooperative groups
    void* args1[] = { &a_ptr, &N };
    hipLaunchCooperativeKernel((void*)decompose_cholesky_mm_kernel_device, gridSize, blockSize, args1, 0, 0);
    hipDeviceSynchronize();


    // printf("AFTER decompose\n");
    // a_cpu = a.cpu();  
    // // Ensure it is contiguous on the CPU
    // a_contig_cpu = a_cpu.contiguous();
    // a_ptr_cpu = a_contig_cpu.data_ptr<float>();
    // for (int i = 0; i < 3; ++i) {
    //     for (int j = 0; j < 3; ++j) {
    //         float value = a_ptr_cpu[i * 3 + j];
    //         printf("A[%d, %d] = %f\n", i, j, value);
    //     }
    // }
    // inverse_lower_mm_kernel_device<<< 1, N>>>(a_ptr, result_ptr, N);
    // Kernel launch with cooperative groups
    void* args2[] = { &a_ptr, &result_ptr, &N };
    hipLaunchCooperativeKernel((void*)inverse_lower_mm_kernel_device, gridSize, blockSize, args2, 0, 0);
    hipDeviceSynchronize();
    // printf("AFTER inverse\n");
    // a_cpu = a.cpu();  
    // // Ensure it is contiguous on the CPU
    // a_contig_cpu = a_cpu.contiguous();
    // a_ptr_cpu = a_contig_cpu.data_ptr<float>();
    // for (int i = 0; i < 3; ++i) {
    //     for (int j = 0; j < 3; ++j) {
    //         float value = a_ptr_cpu[i * 3 + j];
    //         printf("A[%d, %d] = %f\n", i, j, value);
    //     }
    // }
    // multiply_lower_mm_kernel_device<<< 1, N>>>(a_ptr, result_ptr, N);
    hipLaunchCooperativeKernel((void*)multiply_lower_mm_kernel_device, gridSize, blockSize, args2, 0, 0);
    hipDeviceSynchronize();
    // printf("AFTER multiply\n");
    // a_cpu = a.cpu();  
    // // Ensure it is contiguous on the CPU
    // a_contig_cpu = a_cpu.contiguous();
    // a_ptr_cpu = a_contig_cpu.data_ptr<float>();
    // for (int i = 0; i < 3; ++i) {
    //     for (int j = 0; j < 3; ++j) {
    //         float value = a_ptr_cpu[i * 3 + j];
    //         printf("A[%d, %d] = %f\n", i, j, value);
    //     }
    // }

    return result;

}


TORCH_LIBRARY_IMPL(cola_kernels, CUDA, m) {
  m.impl("fuse_kernel_1", &fuse_kernel_1_cuda);
  m.impl("fuse_kernel_2", &fuse_kernel_2_cuda);
}

}

