#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <vector>
#include <iostream>

#include <time.h>
#include <math.h>


#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <stdexcept>
#include <cstdio>
#include <iomanip>

namespace cg = cooperative_groups;

namespace cola_kernels
{

    __global__ void fuse_kernel_1(int numel, const float *a, const float *b, const float *c, float *result)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < numel)
            result[idx] = a[idx] * b[idx] + c[idx];
    }

    at::Tensor fuse_kernel_1_cuda(const at::Tensor &a, const at::Tensor &b, const at::Tensor &c)
    {
        TORCH_CHECK(a.sizes() == b.sizes());
        TORCH_CHECK(a.sizes() == c.sizes());
        TORCH_CHECK(a.dtype() == at::kFloat);
        TORCH_CHECK(b.dtype() == at::kFloat);
        TORCH_CHECK(c.dtype() == at::kFloat);
        TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(b.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(c.device().type() == at::DeviceType::CUDA);
        at::Tensor a_contig = a.contiguous();
        at::Tensor b_contig = b.contiguous();
        at::Tensor c_contig = c.contiguous();
        at::Tensor result = torch::empty(a_contig.sizes(), a_contig.options());
        const float *a_ptr = a_contig.data_ptr<float>();
        const float *b_ptr = b_contig.data_ptr<float>();
        const float *c_ptr = c_contig.data_ptr<float>();
        float *result_ptr = result.data_ptr<float>();

        int numel = a_contig.numel();
        fuse_kernel_1<<<(numel + 255) / 256, 256>>>(numel, a_ptr, b_ptr, c_ptr, result_ptr);
        return result;
    }

#define Index(i, j, N) (j * N + i)


    __global__ void decompose_cholesky(float *a, int N) {
        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
        cg::grid_group grid = cg::this_grid();

        for (int k = 0; k < N; k++) {
            if (thread_id == 0) {
                a[Index(k, k, N)] = sqrt(a[Index(k, k, N)]);
            }
            grid.sync();

            int i = k + 1 + thread_id;
            if(i < N) {
                a[Index(i, k, N)] /= a[Index(k, k, N)];
            }
            grid.sync();

            // Update the submatrix
            if(i < N) {
                for (int j = i; j < N; j++) {
                    a[Index(j, i, N)] -= a[Index(i, k, N)] * a[Index(j, k, N)];
                }
            }
            grid.sync();
        }

        if (thread_id < N) {
            for (int j = thread_id + 1; j < N; j++) {
                a[Index(thread_id, j, N)] = 0.0f;
            }
        }
    }

    __global__ void inverse_lower(float *a, float *aInv, int N)
    {

        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
        cg::grid_group grid = cg::this_grid();

        // Compute the elements of the lower inverse matrix
        for (int j = 0; j < N; j++)
        {
            for (int i = j + 1; i < N; i++)
            {

                if (thread_id == 0)
                {
                    aInv[Index(i, j, N)] = -a[Index(i, j, N)] /
                                            (a[Index(j, j, N)] * a[Index(i, i, N)]);
                }
                grid.sync();

                int k = thread_id + j + 1;
                if (k < i)
                {
                    atomicAdd((float *)&aInv[Index(i, j, N)],
                              -a[Index(i, k, N)] * aInv[Index(k, j, N)] / a[Index(i, i, N)]);
                }
            }
        }
        grid.sync();

        if (thread_id < N) {
            aInv[Index(thread_id, thread_id, N)] = 1.0 / a[Index(thread_id, thread_id, N)];
        }
    }

    __global__ void multiply_lower(float *a, float *aInv, int N)
    {

        int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
        cg::grid_group grid = cg::this_grid();

        // Perform multiplication directly in aInv
        for (int j = 0; j < N; j++)
        {
            for (int i = j; i < N; i++)
            {

                if (thread_id == 0)
                {
                    aInv[Index(i, j, N)] *= aInv[Index(i, i, N)];
                }
                grid.sync();

                int k = thread_id + i + 1;
                if (k < N)
                {
                    atomicAdd((float *)&aInv[Index(i, j, N)],
                              aInv[Index(k, j, N)] * aInv[Index(k, i, N)]);
                }
            }
        }
        grid.sync();


        if (thread_id < N) {
            for (int j = 0; j <= thread_id; j++) {
                aInv[Index(j, thread_id, N)] = aInv[Index(thread_id, j, N)];
            }
        }
    }

    at::Tensor fuse_kernel_2_cuda(at::Tensor &a)
    {
        TORCH_CHECK(a.dtype() == at::kFloat);
        TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
        at::Tensor a_contig = a.contiguous();
        at::Tensor result = torch::zeros(a_contig.sizes(), a_contig.options());
        float *a_ptr = a_contig.data_ptr<float>();
        float *result_ptr = result.data_ptr<float>();

        int N = static_cast<int>(a.size(0));

        dim3 blockSize(256);                                
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x); 
        // Kernel launch with cooperative groups
        void *args1[] = {&a_ptr, &N};
        hipLaunchCooperativeKernel((void *)decompose_cholesky, gridSize, blockSize, args1, 0, 0);
        hipDeviceSynchronize();

        void *args2[] = {&a_ptr, &result_ptr, &N};
        hipLaunchCooperativeKernel((void *)inverse_lower, gridSize, blockSize, args2, 0, 0);
        hipDeviceSynchronize();

        hipLaunchCooperativeKernel((void *)multiply_lower, gridSize, blockSize, args2, 0, 0);
        hipDeviceSynchronize();

        return result;
    }


    __global__ void computeSquareRoot(float *s_ptr, const float *eigenvalues_ptr, int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            float lambda = eigenvalues_ptr[idx];
            s_ptr[idx] = sqrtf(fmaxf(lambda, 0.0f));
        }
    }

    std::tuple<at::Tensor, at::Tensor, at::Tensor> fuse_kernel_3_cuda(at::Tensor &a)
    {

        TORCH_CHECK(a.dtype() == at::kFloat);
        TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);

        at::Tensor a_contig = a.contiguous();
        int m = a_contig.size(0);
        int n = a_contig.size(1);
        int lda = m;

        float *a_ptr = a_contig.data_ptr<float>();


        at::Tensor u = torch::empty({m, m}, a_contig.options());
        at::Tensor s = torch::empty({std::min(m, n)}, a_contig.options());
        at::Tensor vt = torch::empty({n, n}, a_contig.options());
        float *u_ptr = u.data_ptr<float>();
        float *s_ptr = s.data_ptr<float>();
        float *vt_ptr = vt.data_ptr<float>();

        // cuBLAS and cuSOLVER handles
        hipblasHandle_t cublasH;
        hipblasCreate(&cublasH);
        hipsolverHandle_t cusolverH;
        hipsolverDnCreate(&cusolverH);

        float alpha = 1.0f, beta = 0.0f;
        at::Tensor aat = torch::empty({m, m}, a_contig.options());
        at::Tensor ata = torch::empty({n, n}, a_contig.options());
        float *aat_ptr = aat.data_ptr<float>();
        float *ata_ptr = ata.data_ptr<float>();

        hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, m, m, n, &alpha, a_ptr, lda, a_ptr, lda, &beta, aat_ptr, m);
        hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m, &alpha, a_ptr, lda, a_ptr, lda, &beta, ata_ptr, n);

        // U (using A A^T)
        at::Tensor u_eigenvalues = torch::empty({m}, a_contig.options());
        float *u_eigenvalues_ptr = u_eigenvalues.data_ptr<float>();
        int u_lwork = 0;
        hipsolverDnSsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                    m, aat_ptr, m, u_eigenvalues_ptr, &u_lwork);

        float *u_work;
        hipMalloc(&u_work, u_lwork * sizeof(float));
        int *devInfo;
        hipMalloc(&devInfo, sizeof(int));
        hipsolverDnSsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                         m, aat_ptr, m, u_eigenvalues_ptr, u_work, u_lwork, devInfo);


        // V (using A^T A)
        at::Tensor v_eigenvalues = torch::empty({n}, a_contig.options());
        float *v_eigenvalues_ptr = v_eigenvalues.data_ptr<float>();
        int v_lwork = 0;
        hipsolverDnSsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                                    n, ata_ptr, n, v_eigenvalues_ptr, &v_lwork);

        float *v_work;
        hipMalloc(&v_work, v_lwork * sizeof(float));
        hipsolverDnSsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER,
                         n, ata_ptr, n, v_eigenvalues_ptr, v_work, v_lwork, devInfo);


        // Sigma
        int size = std::min(m, n);
        int blockSize = 256;
        int gridSize = (size + blockSize - 1) / blockSize;
        computeSquareRoot<<<gridSize, blockSize>>>(s_ptr, u_eigenvalues_ptr, size);


        // Cleanup
        hipFree(u_work);
        hipFree(v_work);
        hipFree(devInfo);
        hipblasDestroy(cublasH);
        hipsolverDnDestroy(cusolverH);

        return std::make_tuple(aat, s, ata);
    }

#define index(i, j, N) ((i) * (N)) + (j)

    __global__ void sum(float *array, float *out, int size)
    {
        __shared__ float sharedMem[1024]; // Shared memory for block reduction

        int tid = threadIdx.x;
        int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

        sharedMem[tid] = (globalIdx < size) ? array[globalIdx] : 0.0;
        __syncthreads();
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
        {
            if (tid < stride)
            {
                sharedMem[tid] += sharedMem[tid + stride];
            }
            __syncthreads();
        }
        if (tid == 0)
        {
            out[blockIdx.x] = sharedMem[0];
        }
    }

    __global__ void sum_of_squares(float *array, float *out, int size)
    {
        __shared__ float sharedMem[1024]; // Shared memory for block reduction

        int tid = threadIdx.x;
        int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
        sharedMem[tid] = (globalIdx < size) ? array[globalIdx] * array[globalIdx] : 0.0;
        __syncthreads();

        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
        {
            if (tid < stride)
            {
                sharedMem[tid] += sharedMem[tid + stride];
            }
            __syncthreads();
        }
        if (tid == 0)
        {
            out[blockIdx.x] = sharedMem[0];
        }
    }

    __global__ void norm_cal(float *v, float norm, int N)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < N)
        {
            v[tid] /= norm;
        }
    }

    __global__ void update_arr(float *Q, float *V, int N, int i)
    {
        int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (row_idx < N)
        {
            Q[index(i, row_idx, N)] = V[row_idx];
        }
    }

    __global__ void matrix_vector_mul(float *A, float *B, float *C, int N, int i)
    {
        int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (row_idx < N)
        {
            float dot_product = 0.0;
            for (int col_idx = 0; col_idx < N; col_idx++)
            {
                dot_product += A[index(row_idx, col_idx, N)] * B[index(i, col_idx, N)];
            }
            C[row_idx] = dot_product;
        }
    }

    __global__ void update_new_vector(float A, float *B, float *C, int N, int i)
    {
        int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (row_idx < N)
        {
            C[row_idx] -= A * B[index(i,row_idx, N)];
        }
    }

    __global__ void angle_cal(float *Q, float *V, float *ang, int N, int i)
    {
        int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (row_idx < N)
        {
            ang[row_idx] = Q[index(i, row_idx, N)] * V[row_idx];
        }
    }

    std::tuple<at::Tensor, at::Tensor> fuse_kernel_4_cuda(at::Tensor &a, int64_t max_iters=100)
    {

        TORCH_CHECK(a.dtype() == at::kFloat);
        TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
        at::Tensor a_contig = a.contiguous();
        float *A_d = a_contig.data_ptr<float>();

        // int max_iters = 100;
        double diff_t;
        int idx = 0;
        float norm, angle;
        srand(21);
        int random_number = rand();
        int N = a.sizes()[0];
        int blocks = (N + 1023) / 1024;
        float tol = 0.0000001;
        float limit = 1;

        // cpu data
        float *new_vector, *new_vector_r;
        new_vector = (float *)calloc(N, sizeof(float));
        new_vector_r = (float *)calloc(blocks, sizeof(float));

        // cuda data
        float *new_vector_d, *new_vector_d_r, *H_d, *Q_d, *h_vec_d, *ang_d;
        hipMalloc((void **)&new_vector_d, N * sizeof(float));
        hipMalloc((void **)&new_vector_d_r, blocks * sizeof(float));
        hipMalloc((void **)&H_d, (max_iters + 1) * max_iters * sizeof(float));
        hipMalloc((void **)&Q_d, N * (max_iters + 1) * sizeof(float));
        hipMalloc((void **)&h_vec_d, (max_iters + 1) * sizeof(float));
        hipMalloc((void **)&ang_d, N * sizeof(float));

        for (int i = 0; i < N; i++)
        {
            new_vector[i] = ((float)rand() / RAND_MAX);
        }
        // cuda mem copy and set
        hipMemcpy(new_vector_d, new_vector, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemset(H_d, 0, max_iters * (max_iters + 1) * sizeof(float));
        hipMemset(Q_d, 0, N * (max_iters + 1) * sizeof(float));

        // norm calcuations
        sum_of_squares<<<blocks, 1024>>>(new_vector_d, new_vector_d_r, N);
        hipMemcpy(new_vector_r, new_vector_d_r, blocks * sizeof(float), hipMemcpyDeviceToHost);
        norm = 0;
        for (int i = 0; i < blocks; i++)
        {
            norm = norm + new_vector_r[i];
        }
        norm = sqrt(norm);
        norm_cal<<<blocks, 1024>>>(new_vector_d, norm, N);

        // update Q
        update_arr<<<blocks, 1024>>>(Q_d, new_vector_d, N, 0);

        // iterations
        while (idx < max_iters && norm > tol * limit)
        {
            matrix_vector_mul<<<blocks, 1024>>>(A_d, Q_d, new_vector_d, N, idx);
            hipMemset(h_vec_d, 0, (max_iters + 1) * sizeof(float));
            for (int j = 0; j < idx + 1; j++)
            {
                hipMemset(ang_d, 0, N * sizeof(float));
                hipMemset(new_vector_d_r, 0, blocks * sizeof(float));
                memset(new_vector_r, 0, blocks * sizeof(float));
                angle_cal<<<blocks, 1024>>>(Q_d, new_vector_d, ang_d, N, j);
                sum<<<blocks, 1024>>>(ang_d, new_vector_d_r, N);
                hipMemcpy(new_vector_r, new_vector_d_r, blocks * sizeof(float), hipMemcpyDeviceToHost);
                angle = 0;
                for (int i = 0; i < blocks; i++)
                {
                    angle = angle + new_vector_r[i];
                }
                hipMemcpy(&h_vec_d[j], &angle, sizeof(float), hipMemcpyHostToDevice);
                update_new_vector<<<blocks, 1024>>>(angle, Q_d, new_vector_d, N, j);
            }
            hipMemset(new_vector_d_r, 0, blocks * sizeof(float));
            memset(new_vector_r, 0, blocks * sizeof(float));
            sum_of_squares<<<blocks, 1024>>>(new_vector_d, new_vector_d_r, N);
            hipMemcpy(new_vector_r, new_vector_d_r, blocks * sizeof(float), hipMemcpyDeviceToHost);
            norm = 0;
            for (int i = 0; i < blocks; i++)
            {
                norm = norm + new_vector_r[i];
            }
            norm = sqrt(norm);
            if (std::abs(norm) < tol / 2.0)
            {
                norm = tol / 2.0;
            }
            norm_cal<<<blocks, 1024>>>(new_vector_d, norm, N);
            hipMemcpy(&h_vec_d[idx + 1], &norm, sizeof(float), hipMemcpyHostToDevice);
            update_arr<<<((max_iters + 1) + 1023) / 1024, 1024>>>(H_d, h_vec_d, (max_iters + 1), idx);
            update_arr<<<blocks, 1024>>>(Q_d, new_vector_d, N, (idx + 1));
            hipMemcpy(&limit, &H_d[index(0, 0, max_iters)], sizeof(float), hipMemcpyDeviceToHost);
            idx += 1;
        }
        torch::Tensor rH = torch::from_blob(H_d, {(max_iters + 1) * max_iters}, torch::kFloat).to(torch::Device(torch::kCUDA, 1));
        torch::Tensor rQ = torch::from_blob(Q_d, {N * (max_iters + 1)}, torch::kFloat).to(torch::Device(torch::kCUDA, 1));

        hipFree(new_vector_d);
        hipFree(new_vector_d_r);
        hipFree(H_d);
        hipFree(Q_d);
        hipFree(h_vec_d);
        hipFree(ang_d);
        return std::make_tuple(rH, rQ);
    }

    // Error checking macro for CUDA calls
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            hipDeviceReset(); \
            throw std::runtime_error("CUDA error"); \
        } \
    } while (0)

// Error checking macro for cuBLAS calls
#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS error at %s:%d\n", __FILE__, __LINE__); \
            hipDeviceReset(); \
            throw std::runtime_error("cuBLAS error"); \
        } \
    } while (0)

// Constants for CUDA kernel configuration
constexpr int BLOCK_SIZE = 256;     // Number of threads per block
constexpr int MAX_BLOCKS = 65535;   // Maximum number of blocks

// Initialize CUDA random number generator states
__global__ void setup_curand_kernel(hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(clock64(), idx, 0, &state[idx]);
}

// Generate random vectors for Hutchinson estimation
__global__ void generate_random_vector(hiprandState *state, float *z, int n, int bs, bool is_rademacher) {
    // Each thread gets its own random state in shared memory
    __shared__ hiprandState localState[BLOCK_SIZE];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Copy state to shared memory for faster access
    localState[threadIdx.x] = state[tid];
    
    // Generate random values for the entire matrix
    int total_elements = n * bs;
    for (int i = tid; i < total_elements; i += gridDim.x * blockDim.x) {
        float rand_val = hiprand_normal(&localState[threadIdx.x]);
        // Either Rademacher (+1/-1) or normal distribution
        z[i] = is_rademacher ? ((rand_val >= 0.0f) ? 1.0f : -1.0f) : rand_val;
    }
    
    // Save state back to global memory
    state[tid] = localState[threadIdx.x];
}

__global__ void compute_diagonal_estimate(
    const float* __restrict__ Az,     // Matrix-vector product Az
    const float* __restrict__ z,      // Random vector z
    float* __restrict__ diag_sum,     // Running sum of diagonal estimates
    float* __restrict__ diag_sumsq,   // Running sum of squares for variance estimation
    int n,                           // Matrix dimension
    int bs,                          // Batch size for random vectors
    int k,                           // Diagonal offset (0 for main diagonal)
    int iter                         // Current iteration number
) {
    // Shared memory for temporary storage of sums and squared sums
    __shared__ float shared_sum[BLOCK_SIZE];
    __shared__ float shared_sumsq[BLOCK_SIZE];
    
    // Calculate thread ID and grid stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Process diagonal elements in strided fashion
    for (int i = tid; i < n - abs(k); i += stride) {
        // Initialize accumulators for this diagonal element
        shared_sum[threadIdx.x] = 0.0f;
        shared_sumsq[threadIdx.x] = 0.0f;
        
        // Process batch of random vectors
        #pragma unroll 4
        for (int b = 0; b < bs; b++) {
            int idx = i + (k >= 0 ? k : 0);  // Adjust index for off-diagonal elements
            float est = Az[i + b * n] * z[idx + b * n];  // Compute estimate for this batch
            shared_sum[threadIdx.x] += est;              // Accumulate sum
            shared_sumsq[threadIdx.x] += est * est;      // Accumulate sum of squares
        }
        
        // Normalize by batch size
        shared_sum[threadIdx.x] /= bs;
        shared_sumsq[threadIdx.x] /= bs;
        
        __syncthreads();
        
        // Update running statistics
        if (iter == 0) {
            // First iteration: just store the values
            diag_sum[i] = shared_sum[threadIdx.x];
            diag_sumsq[i] = shared_sumsq[threadIdx.x];
        } else {
            // Subsequent iterations: update mean and sum of squares
            float old_mean = diag_sum[i];
            float delta = shared_sum[threadIdx.x] - old_mean;
            diag_sum[i] += delta / (iter + 1);                    // Update mean
            diag_sumsq[i] += delta * (shared_sum[threadIdx.x] - diag_sum[i]); // Update sum of squares
        }
    }
}

__global__ void compute_relative_error_kernel(
    const float* diag_sum,      // Running sum of diagonal estimates
    const float* diag_sumsq,    // Running sum of squares for variance estimation
    float* max_error,           // Output: maximum relative error across all elements
    int n,                      // Number of diagonal elements
    int total_iters            // Total number of iterations completed
) {
    // Shared memory for block-wise reduction of maximum error
    __shared__ float shared_max_error[BLOCK_SIZE];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize local maximum error for this thread
    float local_max_error = 0.0f;
    
    // Process elements with grid-stride loop
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        float mean = diag_sum[i];
        float variance = diag_sumsq[i] / (total_iters - 1);
        float stderr = sqrtf(variance / total_iters);
        // Relative error with minimum denominator of 0.1 to avoid division by zero
        float rel_error = stderr / fmaxf(fabsf(mean), 0.1f);
        local_max_error = fmaxf(local_max_error, rel_error);
    }
    
    // Store local maximum in shared memory
    shared_max_error[threadIdx.x] = local_max_error;
    __syncthreads();
    
    // Parallel reduction to find maximum error within the block
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared_max_error[threadIdx.x] = fmaxf(
                shared_max_error[threadIdx.x],
                shared_max_error[threadIdx.x + s]
            );
        }
        __syncthreads();
    }
    
    // First thread in block updates global maximum using atomic operation
    if (threadIdx.x == 0) {
        atomicMax((int*)max_error, __float_as_int(shared_max_error[0]));
    }
}

    at::Tensor fuse_kernel_5_cuda(
        at::Tensor &mat,      // Input matrix (host)
        at::Tensor &diag,          // Output diagonal (host)
        int64_t bs = 100,            // Batch size
        double tol = 3e-2f,       // Tolerance
        int64_t max_iters = 10000,   // Maximum iterations
        int64_t k = 0,               // Diagonal offset
        bool use_rademacher = false  // Use rademacher instead of normal distribution
    ) {

        TORCH_CHECK(mat.dtype() == at::kFloat);
        TORCH_INTERNAL_ASSERT(mat.device().type() == at::DeviceType::CUDA);
        TORCH_CHECK(diag.dtype() == at::kFloat);
        TORCH_INTERNAL_ASSERT(diag.device().type() == at::DeviceType::CUDA);


        at::Tensor mat_contig = mat.contiguous();
        at::Tensor diag_contig = diag.contiguous();

        int n = mat_contig.size(0); // Matrix dimension

        float* matrix = mat_contig.data_ptr<float>();
        float* diagonal = diag_contig.data_ptr<float>();


    // Create CUDA stream for asynchronous operations
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));
        
        // Initialize cuBLAS handle and set stream
        hipblasHandle_t handle;
        CUBLAS_CHECK(hipblasCreate(&handle));
        CUBLAS_CHECK(hipblasSetStream(handle, stream));
        
        // Custom deleter for CUDA memory management
        struct CudaDeleter {
            void operator()(void* p) { hipFree(p); }
        };
        // printf("CUDAstream created\n");
        // Smart pointers for CUDA memory management
        std::unique_ptr<float, CudaDeleter> d_matrix;      // Device matrix
        std::unique_ptr<float, CudaDeleter> d_z;           // Random vectors
        std::unique_ptr<float, CudaDeleter> d_Az;          // Matrix-vector products
        std::unique_ptr<float, CudaDeleter> d_diag_sum;    // Running sum of estimates
        std::unique_ptr<float, CudaDeleter> d_diag_sumsq;  // Running sum of squares
        std::unique_ptr<hiprandState, CudaDeleter> d_rand_state;  // RNG states
        
        float *ptr;
        
        // Allocate device memory
        CUDA_CHECK(hipMalloc(&ptr, n * n * sizeof(float)));
        d_matrix.reset(ptr);
        CUDA_CHECK(hipMalloc(&ptr, n * bs * sizeof(float)));
        d_z.reset(ptr);
        CUDA_CHECK(hipMalloc(&ptr, n * bs * sizeof(float)));
        d_Az.reset(ptr);
        CUDA_CHECK(hipMalloc(&ptr, (n - abs(k)) * sizeof(float)));
        d_diag_sum.reset(ptr);
        CUDA_CHECK(hipMalloc(&ptr, (n - abs(k)) * sizeof(float)));
        d_diag_sumsq.reset(ptr);
        

        // Allocate RNG states
        hiprandState *rand_ptr;
        CUDA_CHECK(hipMalloc(&rand_ptr, BLOCK_SIZE * sizeof(hiprandState)));
        d_rand_state.reset(rand_ptr);
        
        // Copy matrix to device and initialize accumulators
        // CUDA_CHECK(hipMemcpyAsync(d_matrix.get(), matrix, n * n * sizeof(float),
        //                         hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_matrix.get(), matrix, n * n * sizeof(float),
                                hipMemcpyDeviceToDevice, stream));
        CUDA_CHECK(hipMemsetAsync(d_diag_sum.get(), 0, (n - abs(k)) * sizeof(float), stream));
        CUDA_CHECK(hipMemsetAsync(d_diag_sumsq.get(), 0, (n - abs(k)) * sizeof(float), stream));
        
        // Configure kernel grid
        int num_blocks = std::min((n + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_BLOCKS);
        setup_curand_kernel<<<num_blocks, BLOCK_SIZE, 0, stream>>>(d_rand_state.get());
        
        // cuBLAS matrix multiplication constants
        const float alpha = 1.0f;
        const float beta = 0.0f;
        
        // Convergence control variables
        float rel_error = tol + 1.0f;
        float *d_max_error;
        CUDA_CHECK(hipMalloc(&d_max_error, sizeof(float)));
        
        // Main iteration loop
        int iter;
        for (iter = 0; iter < max_iters && rel_error > tol; iter++) {
            // printf("Iter: %d\n", iter);
            // Generate batch of random vectors
            generate_random_vector<<<num_blocks, BLOCK_SIZE, 0, stream>>>(
                d_rand_state.get(), d_z.get(), n, bs, use_rademacher);
            
            // Compute matrix-vector products using cuBLAS
            CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                    n, bs, n,
                                    &alpha,
                                    d_matrix.get(), n,
                                    d_z.get(), n,
                                    &beta,
                                    d_Az.get(), n));
            
            // Update diagonal estimates
            compute_diagonal_estimate<<<num_blocks, BLOCK_SIZE, 0, stream>>>(
                d_Az.get(), d_z.get(), d_diag_sum.get(), d_diag_sumsq.get(), 
                n, bs, k, iter);
            
            // Check convergence every 10 iterations
            if ((iter + 1) % 10 == 0 && iter > 0) {
                CUDA_CHECK(hipMemsetAsync(d_max_error, 0, sizeof(float), stream));
                compute_relative_error_kernel<<<num_blocks, BLOCK_SIZE, 0, stream>>>(
                    d_diag_sum.get(), d_diag_sumsq.get(), d_max_error, 
                    n - abs(k), iter + 1);
                CUDA_CHECK(hipMemcpyAsync(&rel_error, d_max_error, sizeof(float),
                                        hipMemcpyDeviceToHost, stream));
                CUDA_CHECK(hipStreamSynchronize(stream));
            }
        }
        
        // Copy results back to host
        // CUDA_CHECK(hipMemcpyAsync(diagonal, d_diag_sum.get(),
        //                         (n - abs(k)) * sizeof(float), 
        //                         hipMemcpyDeviceToHost, stream));
        CUDA_CHECK(hipMemcpyAsync(diagonal, d_diag_sum.get(),
                        (n - abs(k)) * sizeof(float), 
                        hipMemcpyDeviceToDevice, stream));
        CUDA_CHECK(hipStreamSynchronize(stream));
        
        // Print convergence information
        // std::cout << "Completed after " << iter << " iterations\n";
        // std::cout << "Final relative error: " << rel_error << "\n";
        
        // Cleanup
        CUDA_CHECK(hipFree(d_max_error));
        CUDA_CHECK(hipStreamDestroy(stream));
        CUBLAS_CHECK(hipblasDestroy(handle));

        return diag;
    }

    TORCH_LIBRARY_IMPL(cola_kernels, CUDA, m)
    {
        m.impl("fuse_kernel_1", &fuse_kernel_1_cuda);
        m.impl("fuse_kernel_2", &fuse_kernel_2_cuda);
        m.impl("fuse_kernel_3", &fuse_kernel_3_cuda);
        m.impl("fuse_kernel_4", &fuse_kernel_4_cuda);
        m.impl("fuse_kernel_5", &fuse_kernel_5_cuda);
    }

}
